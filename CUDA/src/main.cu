#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "float.h"
#include <hip/hip_runtime_api.h>
#include <hiprand_uniform.h>

#include "vec3.cuh"
#include "ray.cuh"
#include "sphere.cuh"
#include "hitable_list.cuh"
#include "camera.cuh"
#include "material.cuh"
#include "utility.cuh"

#include <iostream>
#include <sys/time.h>
#include <fstream>

/* Auxiliary Functions */

// Utility function to get current CPU time in seconds
double cpuSeconds() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

// Device function to recursively calculate the color of a ray
__device__ vec3 color(const ray& r, hitable **world, int depth, int max_depth, hiprandState* rand_state) {
    hit_record rec;
    if ((*world)->hit(r, 0.001, MAXFLOAT, rec)) { 
        ray scattered;
        vec3 attenuation;
        if (depth < max_depth && rec.mat_ptr->scatter(r, rec, attenuation, scattered, rand_state)) {
             return attenuation*color(scattered, world, depth+1, max_depth, rand_state);
        }
        else {
            return vec3(0,0,0);
        }
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5*(unit_direction.y() + 1.0);
        return (1.0-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
}

// Device function to create a peudo-random world scene defined in GPU memory
__device__ hitable* random_scene(int number_grid, hitable **list, hiprandState *random_state) {
    list[0] =  new sphere(vec3(0,-1000,0), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));
    int i = 1;
    float offset[] = {0.1, -0.42, 0.17, -0.27, 0.29, 0.17, -0.35, 0.11, -0.29};
    float color_offset[] = {0.1, 0.54, 0.87, 0.34, 0.75, 0.18, 0.99, 0.48, 0.74};
    int color_length = 9;
    for (int a = -1*number_grid; a < number_grid; a++) {
        for (int b = -1*number_grid; b < number_grid; b++) {
            vec3 center(a + offset[i%9],0.2,b - offset[i%9]);
            //if ((center-vec3(4,0.2,0)).length() > 0.9) {
                if (i%3==0) {  // diffuse
                    list[i++] = new sphere(center, 0.2 + i%3*0.1, new lambertian(vec3(color_offset[i%color_length], color_offset[(i+2)%color_length], color_offset[(i+5)%color_length])));
                }
                else if (i%3==1) { // metal
                    list[i++] = new sphere(center, 0.2 + i%3*0.1,
                            new metal(vec3(color_offset[(i+3)%color_length], color_offset[i%color_length], color_offset[(i+8)%color_length]),  color_offset[(i+5)%color_length]));
                }
                else {  // glass
                    list[i++] = new sphere(center, 0.2 + i%3*0.1, new dielectric(1.5));
                }
            //}
        }
    }

    list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
    list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
    list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));

    return new hitable_list(list, 4*number_grid*number_grid+4);
}

/* CUDA Kernel Functions */

// Kernel functon to create cuda random states foreach launched thread,
// each has a unqiue seed
__global__ void random_render_init(int width, int height, hiprandState *random_state_list) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= width) || (j >= height)) return;
	int pixel_index = j*width + i;
	hiprand_init(1984+pixel_index, 0, 0, &random_state_list[pixel_index]);
}

// Kernel function to create world scene and camera defined in GPU memory
__global__ void create_world(int width, int height, int number_grid, hitable **list, camera **world_camera, hitable **world, hiprandState *random_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hitable **temp_list = new hitable*[5];
		temp_list[0] = new sphere(vec3(0,0,-1), 0.5, new lambertian(vec3(0.1, 0.2, 0.5)));
		temp_list[1] = new sphere(vec3(0,-100.5,-1), 100, new lambertian(vec3(0.8, 0.8, 0.0)));
		temp_list[2] = new sphere(vec3(1,0,-1), 0.5, new metal(vec3(0.8, 0.6, 0.2), 0.0));
		temp_list[3] = new sphere(vec3(-1,0,-1), 0.5, new dielectric(1.5));
		temp_list[4] = new sphere(vec3(-1,0,-1), -0.45, new dielectric(1.5));
		*world = new hitable_list(temp_list,5);
		*world = random_scene(number_grid, list, random_state);
		
		vec3 lookfrom(13,2,3);
		vec3 lookat(0,0,0);
		float dist_to_focus = 10.0;
		float aperture = 0.1;
		*world_camera = new camera(lookfrom, lookat, vec3(0,1,0), 20, float(width)/float(height), aperture, dist_to_focus);
	}
}

// Kernel function to render all rays of a pixel
__global__ void render(vec3 *frame_buffer, int width, int height, int samples, int max_depth, camera **world_camera, hitable **world, hiprandState *random_state_list) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= width) || (j >= height)) return;
	int pixel_index = j*width + i;
	hiprandState current_random_state = random_state_list[pixel_index];
	vec3 col(0, 0, 0);
	for (int k = 0; k < samples; k++) {
		float u = float(i + random_double(&current_random_state)) / float(width);
		float v = float(j + random_double(&current_random_state)) / float(height);
		ray r = (*world_camera)->get_ray(u, v, &current_random_state);
		vec3 p = r.point_at_parameter(2.0);
		col += color(r, world,0, max_depth, &current_random_state);
	}
	random_state_list[pixel_index] = current_random_state; 
	col /= float(samples);
	col = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));
	frame_buffer[pixel_index] = col;
}

// Kernel function to free the memory defined in GPU memory
__global__ void free_world(int hitable_number, hitable **list, camera **world_camera, hitable **world) {
	for (int i = 0; i < hitable_number; i++) {
		delete ((sphere *)list[i])->mat_ptr;
        delete list[i];
	}
	delete *world;
	delete *world_camera;
}

int main(int argc, char* argv[]) {
    int nx = 1200;				// Number of pixels in x direction
    int ny = 800; 				// Number of pixels in y direction
    int ns = 10; 				// Number of samples per pixel
    int max_depth = 50; 		// Number of reflections
    int number_grid = 11; 		// Number of grid points for little spheres

    if (argc > 1) {
        if (argc != 6) {
            std::cout << "Error in input parameters. \n Shoul be file path, width, samples per pixel, max depth, number of grid points\n" << std::endl;
            return 1;
        }
        nx = std::atoi(argv[2]);
        ny = float(nx) * 0.5625;
        ns = std::atoi(argv[3]);
        max_depth = std::atoi(argv[4]);
        number_grid = std::atoi(argv[5]);
    }

	size_t stackSize = 8192;
	hipDeviceSetLimit(hipLimitStackSize, stackSize);

	// CUDA memory allocation
	int pixels = nx * ny;
	vec3* frame_buffer;
	hipMallocManaged(&frame_buffer, pixels * sizeof(vec3));
	checkCudaError("CUDA Malloc frame buffer failed");

	hiprandState* device_random_state;
	hipMalloc(&device_random_state, sizeof(hiprandState));
	checkCudaError("CUDA Malloc random state failed");

	hiprandState* device_random_state_list;
	hipMalloc(&device_random_state_list, pixels * sizeof(hiprandState));
	checkCudaError("CUDA Malloc random state list failed");

    int hitable_number = 4*number_grid*number_grid+4;
	hitable **device_list;
	hipMalloc(&device_list, hitable_number * sizeof(hitable *));
	checkCudaError("CUDA Malloc list failed");

	hitable **device_world;
	hipMalloc(&device_world, sizeof(hitable *));
	checkCudaError("CUDA Malloc world failed");

	camera **device_camera;
	hipMalloc(&device_camera, sizeof(camera *));
	checkCudaError("CUDA Malloc camera failed");

	// Create world kernel launch
	hipDeviceSynchronize();
	checkCudaError("CUDA Sync random world init failed");
	create_world<<<1, 1>>>(nx, ny, number_grid, device_list, device_camera, device_world, device_random_state);
	checkCudaError("CUDA create world kernel failed");
	hipDeviceSynchronize();
	checkCudaError("CUDA Sync create world failed");

    // setup vars for kernel
	int threads = 8;
	dim3 block_size(threads, threads, 1);
	dim3 grid_size(nx/block_size.x + 1, ny/block_size.y + 1, 1);

	double dStart = cpuSeconds();

	// CUDA render kernel calls
	std::clog << "Init" << std::endl;
	random_render_init<<<grid_size, block_size>>>(nx, ny, device_random_state_list);
	checkCudaError("CUDA random render init kernel failed");
	hipDeviceSynchronize();
	checkCudaError("CUDA Sync random render init failed");
	std::clog << "Render" << std::endl;
	render<<<grid_size, block_size>>>(frame_buffer, nx, ny, ns, max_depth, device_camera, device_world, device_random_state_list);
	checkCudaError("CUDA render kernel failed");
	hipDeviceSynchronize();
	checkCudaError("CUDA Sync render failed");
	// Output frame buffer as ppm image
	std::cout << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny-1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			int pixel_index = j*nx+i;
			int ir = int(255.99*frame_buffer[pixel_index].r()); 
			int ig = int(255.99*frame_buffer[pixel_index].g()); 
			int ib = int(255.99*frame_buffer[pixel_index].b()); 
			std::cout << ir << " " << ig << " " << ib << "\n";
		}
	}

	double dElaps = cpuSeconds() - dStart;
	std::clog << "Duration of render: " << dElaps << "s" << std::endl;


	// Free memory
	std::clog << "Free the world" << std::endl;
	free_world<<<1,1>>>(hitable_number, device_list, device_camera, device_world);
	checkCudaError("CUDA free world kernel failed");
	hipDeviceSynchronize();
	checkCudaError("CUDA Sync free world failed");
	hipFree(device_world);
	checkCudaError("CUDA Free world failed");
	hipFree(device_camera);
	checkCudaError("CUDA Free camera failed");
	hipFree(device_list);
	checkCudaError("CUDA Free list failed");
	hipFree(device_random_state);
	checkCudaError("CUDA Free random state failed");
	hipFree(device_random_state_list);
	checkCudaError("CUDA Free random state list failed");
	hipFree(frame_buffer);
	checkCudaError("CUDA Free frame buffer failed");

	hipDeviceReset();

	// Store profiling data
	if (argc > 1) {
		std::fstream file;
		std::string filename = argv[1];
		// Check if file exists
		file.open(filename, std::ios::in);
		bool fileExists = file.is_open();
		file.close();

		// Open the file in append mode
		file.open(filename, std::ios::app);
		if (!fileExists) {
			// If file doesn't exists, write a header first
			file << "Width,Samples,Depth,NumberGrid,Time\n";
		}

		// Append new profiling data to the file
		file << nx << "," << ns << "," << max_depth << "," << number_grid << "," << dElaps << "\n";

		file.close();
	}
}

